
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "GMS_cuBLAS_L1_wrappers.cuh"
#include "GMS_gpu_config.cuh"
#if (PROFILE_HOST_TO_DEVICE) == 1
#include <immintrin.h> //rdtscp
#endif 
#include "GMS_cuda_memops.cuh"

static const uint64_t rdtscp_cost = 42; // Skylake uarch

void
cuBLAS_Isamax_iface(const float * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t * result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr
                    uint64_t * __restrict tsc_delta) { // for profiling usage 
                                  
    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    float * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_float_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(float)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIsamax(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIsamax executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;  
                return;  
      }
}


cuBLAS_Icamax_iface(const hipComplex * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t * result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr
                    uint64_t * __restrict tsc_delta) { // for profiling usage 
                                  
    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
     *err    = 0;
     *ierr   = 0;
     *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    hipComplex * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_complex4_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(hipComplex)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIcamax(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIcamax executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
       return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;  
                return;  
      }
}


void
cuBLAS_Isamin_iface(const float * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t *     __restrict result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr,
                    uint64_t * __restrict tsc_delta) {
  
     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    float * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    cudaError_t_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_float_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(float)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIsamin(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIsamin executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status; 
                return;   
      }
} 


void
cuBLAS_Icamin_iface(const hipComplex * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t *  __restrict result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr,
                    uint64_t * __restrict tsc_delta) {
      
      if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    hipComplex * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_complex4_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(hipComplex)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIcamin(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIcamin executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
       return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;   
                return; 
      }
}


void
cuBLAS_Sasum_iface(const float * __restrict h_ptr,
                   const int32_t n,
                   const int32_t incx,
                   float * __restrict result,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    float * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_float_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(float)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasSasum(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasSasum executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;  
                return;  
      }
}


void
cuBLAS_Scasum_iface(const hipComplex * __restrict h_ptr,
                   const int32_t n,
                   const int32_t incx,
                   hipComplex * __restrict result,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {
    
     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    hipComplex * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    cudaError_t_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_complex4_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(hipComplex)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasScasum(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasScasum executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr));
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;   
                return; 
      }
} 


void
cuBLAS_Sgemm_iface(hipblasOperation_t transa,
                   hipblasOperation_t trnasb,
                   const int32_t m,
                   const int32_t n,
                   const int32_t k,
                   float alpha,
                   const float * __restrict A,
                   const int32_t LDA,
                   const float * __restrict B,
                   const int32_t LDB,
                   float beta,
                   float * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta)  {

    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     float * __restrict   d_ptrA = NULL;
     float * __restrict   d_ptrB = NULL;
     float * __restrict   d_ptrC = NULL;
     float alph = alpha;
     float bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     alloc_float_gpu(d_ptrA,(size_t)(m*k),&merr);
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrB,(size_t)(n*k),&merr);
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrC,(size_t)(m*n),&merr);
     if(merr != 0) goto Error;
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(m*k),
                          hipMemcpyHostToDevice,stream));
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(float)*(n*k),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasSgemm(handle,transa,transb,m,n,k,&alph,
                                        d_ptrA,LDA,d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasSgemm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(float)*(m*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }
}


void
cuBLAS_Cgemm_iface(hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   const int32_t m,
                   const int32_t n,
                   const int32_t k,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   const hipComplex * __restrict B,
                   const int32_t LDB,
                   hipComplex beta,
                   hipComplex * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

   if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     alloc_complex4_gpu(d_ptrA,(size_t)(m*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrB,(size_t)(n*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(m*n),&merr);
     if(merr != 0) goto Error;
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(m*k),
                          hipMemcpyHostToDevice,stream));
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(n*k),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasCgemm(handle,transa,transb,m,n,k,&alph,
                                        d_ptrA,LDA,d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasCgemm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(m*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  
} 



void
cuBLAS_Cgemm3m_iface(hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   const int32_t m,
                   const int32_t n,
                   const int32_t k,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   const hipComplex * __restrict B,
                   const int32_t LDB,
                   hipComplex beta,
                   hipComplex * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

   if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     alloc_complex4_gpu(d_ptrA,(size_t)(m*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrB,(size_t)(n*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(m*n),&merr);
     if(merr != 0) goto Error;
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(m*k),
                          hipMemcpyHostToDevice,stream));
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(n*k),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(cublasCgemm3m(handle,transa,transb,m,n,k,&alph,
                                        d_ptrA,LDA,d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("cublasCgemm3m executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(m*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  
} 



  


