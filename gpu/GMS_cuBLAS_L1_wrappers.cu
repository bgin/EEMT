
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "GMS_cuBLAS_L1_wrappers.cuh"
#include "GMS_gpu_config.cuh"
#if (PROFILE_HOST_TO_DEVICE) == 1
#include <immintrin.h> //rdtscp
#endif 
#include "GMS_cuda_memops.cuh"

static const uint64_t rdtscp_cost = 42; // Skylake uarch

void
cuBLAS_Isamax_iface(const float * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t * result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr
                    uint64_t * __restrict tsc_delta) { // for profiling usage 
                                  
    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    float * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_float_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(float)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIsamax(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIsamax executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;  
                return;  
      }
}


cuBLAS_Icamax_iface(const hipComplex * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t * result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr
                    uint64_t * __restrict tsc_delta) { // for profiling usage 
                                  
    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
     *err    = 0;
     *ierr   = 0;
     *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    hipComplex * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_complex4_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(hipComplex)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIcamax(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIcamax executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
       return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;  
                return;  
      }
}


void
cuBLAS_Isamin_iface(const float * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t *     __restrict result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr,
                    uint64_t * __restrict tsc_delta) {
  
     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    float * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    cudaError_t_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_float_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(float)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIsamin(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIsamin executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status; 
                return;   
      }
} 


void
cuBLAS_Icamin_iface(const hipComplex * __restrict h_ptr,
                    const int32_t n,
                    const int32_t incx,
                    int32_t *  __restrict result,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr,
                    uint64_t * __restrict tsc_delta) {
      
      if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    hipComplex * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_complex4_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(hipComplex)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasIcamin(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasIcamin executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
       return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;   
                return; 
      }
}


void
cuBLAS_Sasum_iface(const float * __restrict h_ptr,
                   const int32_t n,
                   const int32_t incx,
                   float * __restrict result,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    float * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    hipError_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_float_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(float)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasSasum(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasSasum executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr);
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;  
                return;  
      }
}


void
cuBLAS_Scasum_iface(const hipComplex * __restrict h_ptr,
                   const int32_t n,
                   const int32_t incx,
                   hipComplex * __restrict result,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {
    
     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
    *result = 0;
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
    hipComplex * __restrict d_ptr = NULL;
    hipblasHandle_t handle    = NULL;
    hipStream_t   stream    = NULL;
    cudaError_t_t status;
    int32_t merr = 0;
    GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
    GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
    GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
    alloc_complex4_gpu(&d_ptr[0],(size_t)n,&merr);
    if(merr != 0) goto Error;
    GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptr,h_ptr,sizeof(hipComplex)*((size_t)n),
                         hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      dummy1    = __rdtscp(&dummy2);
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
     GMS_CUBLAS_STAT_CHECK(hipblasScasum(handle,n,d_ptr,incx,&result));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasScasum executed in: %llu reference cycles\n",*tsc_delta);
#endif
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptr));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return;
      Error: 
            {
                if(d_ptr) hipFree(d_ptr));
                hipblasDestroy(handle);
                hipStreamDestroy(stream);  
                ierr = -2;
                *err = status;   
                return; 
      }
} 




  


