
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "GMS_cuBLAS_L3_wrappers.cuh"
#include "GMS_gpu_config.cuh"
#if (PROFILE_HOST_TO_DEVICE) == 1
#include <immintrin.h> //rdtscp
#endif 
#include "GMS_cuda_memops.cuh"


    #define GMS_MIN(a,b) (((a)<(b))?(a):(b))

    #define GMS_MAX(a,b) (((a)>(b))?(a):(b))


void
cuBLAS_Sgemm_iface(hipblasOperation_t transa,
                   hipblasOperation_t trnasb,
                   const int32_t m,
                   const int32_t n,
                   const int32_t k,
                   float alpha,
                   const float * __restrict A,
                   const int32_t LDA,
                   const float * __restrict B,
                   const int32_t LDB,
                   float beta,
                   float * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta)  {

    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     float * __restrict   d_ptrA = NULL;
     float * __restrict   d_ptrB = NULL;
     float * __restrict   d_ptrC = NULL;
     float alph = alpha;
     float bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     alloc_float_gpu(d_ptrA,(size_t)(m*k),&merr);
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrB,(size_t)(n*k),&merr);
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrC,(size_t)(m*n),&merr);
     if(merr != 0) goto Error;
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(m*k),
                          hipMemcpyHostToDevice,stream));
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(float)*(n*k),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasSgemm(handle,transa,transb,m,n,k,&alph,
                                        d_ptrA,LDA,d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasSgemm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(float)*(m*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }
}


void
cuBLAS_Cgemm_iface(hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   const int32_t m,
                   const int32_t n,
                   const int32_t k,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   const hipComplex * __restrict B,
                   const int32_t LDB,
                   hipComplex beta,
                   hipComplex * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

   if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     alloc_complex4_gpu(d_ptrA,(size_t)(m*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrB,(size_t)(n*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(m*n),&merr);
     if(merr != 0) goto Error;
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(m*k),
                          hipMemcpyHostToDevice,stream));
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(n*k),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasCgemm(handle,transa,transb,m,n,k,&alph,
                                        d_ptrA,LDA,d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasCgemm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(m*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  
} 



void
cuBLAS_Cgemm3m_iface(hipblasOperation_t transa,
                   hipblasOperation_t transb,
                   const int32_t m,
                   const int32_t n,
                   const int32_t k,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   const hipComplex * __restrict B,
                   const int32_t LDB,
                   hipComplex beta,
                   hipComplex * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

   if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     alloc_complex4_gpu(d_ptrA,(size_t)(m*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrB,(size_t)(n*k),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(m*n),&merr);
     if(merr != 0) goto Error;
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(m*k),
                          hipMemcpyHostToDevice,stream));
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(n*k),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(cublasCgemm3m(handle,transa,transb,m,n,k,&alph,
                                        d_ptrA,LDA,d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("cublasCgemm3m executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(m*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  
} 



void
cuBLAS_Chemm_iface(hipblasSideMode_t side,
                   hipblasFillMode_t uplo,
                   const int32_t m,
                   const int32_t n,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   const hipComplex * __restrict B,
                   const int32_t LDB,
                   const hipComplex beta,
                   hipComplex * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {\

    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(side==HIPBLAS_SIDE_LEFT) {
        alloc_complex4_gpu(d_ptrA,(size_t)(LDA*m),&merr);
     }else{
        alloc_complex4_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrB,(size_t)(LDB*n),&merr);
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(LDC*n),&merr);
     if(merr != 0) goto Error;
     if(side==HIPBLAS_SIDE_LEFT) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*m),
                          hipMemcpyHostToDevice,stream));
     }else {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*n),
                          hipMemcpyHostToDevice,stream));
     } 
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(LDB*n),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasChemm(handle,side,uplo,m,n,&alph,d_ptrA,LDA,
                                        d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasChemm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(LDC*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  


}


void
cuBLAS_Cher2k_iface(hipblasOperation_t transa,
                    hipblasOperation_t transb,
                    hipblasFillMode_t  uplo,
                    const int32_t n,
                    const int32_t k,
                    const hipComplex alpha,
                    const hipComplex * __restrict A,
                    const int32_t LDA,
                    const hipComplex * __restrict B,
                    const int32_t LDB,
                    const hipComplex beta,
                    hipComplex * __restrict C,
                    const int32_t LDC,
                    hipError_t * __restrict err,
                    int32_t * __restrict ierr,
                    uint64_t * __restrict tsc_delta) {

    if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(transa==HIPBLAS_OP_N) {
         alloc_complex4_gpu(d_ptrA,(size_t)(LDA*k),&merr);
     }else{
         alloc_complex4_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     if(transb==HIPBLAS_OP_N) {
        alloc_complex4_gpu(d_ptrB,(size_t)(LDB*k),&merr);
     }
      else {
         alloc_complex4_gpu(d_ptrB,(size_t)(LDB*n),&merr); 
     }
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(LDC*n),&merr);
     if(merr != 0) goto Error;
     if(transa==HIPBLAS_OP_N) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*k),
                          hipMemcpyHostToDevice,stream));
     }else {
         GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*n),
                          hipMemcpyHostToDevice,stream));
     }
     if(transb==HIPBLAS_OP_N) {
         GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(LDB*k),
                          hipMemcpyHostToDevice,stream));
     }else {
         GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(hipComplex)*(LDB*n),
                          hipMemcpyHostToDevice,stream));
     }
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasCher2k(handle,uplo,transa,n,k,&alph,d_ptrA,LDA,
                                         d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasCher2k executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(LDC*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  

} 


void
cuBLAS_Ssymm_iface(hipblasSideMode_t side,
                   hipblasFillMode_t uplo,
                   int32_t m,
                   int32_t n,
                   const float alpha,
                   const float * __restrict A,
                   const int32_t LDA,
                   const float * __restrict B,
                   int32_t LDB,
                   const float beta,
                   float * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict,
                   int32_t * __restrict,
                   uint64_t * __restrict)  {
    
     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     float * __restrict   d_ptrA = NULL;
     float * __restrict   d_ptrB = NULL;
     float * __restrict   d_ptrC = NULL;
     float alph = alpha;
     float bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(side==HIPBLAS_SIDE_LEFT) {
         alloc_float_gpu(d_ptrA,(size_t)(LDA*m),&merr);
     }else{
         alloc_float_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrB,(size_t)(LDB*n),&merr);
    
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrC,(size_t)(LDC*n),&merr);
     if(merr != 0) goto Error;
     if(side==HIPBLAS_SIDE_LEFT) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(LDA*m),
                          hipMemcpyHostToDevice,stream));
      }else {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(LDA*n),
                          hipMemcpyHostToDevice,stream)); 
      }
     GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,B,sizeof(float)*(LDB*n),
                          hipMemcpyHostToDevice,stream));
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasSsymm(handle,side,uplo,m,n,&alph,d_ptrA,LDA,
                                         d_ptrB,LDB,&bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasSsymm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(float)*(LDC*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrB) hipFree(d_ptrB);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  


}


void
cuBLAS_Ssyrk_iface(hipblasFillMode_t uplo,
                   hipblasOperation_t trans,
                   const int32_t n,
                   const int32_t k,
                   const float alpha,
                   const float * __restrict A,
                   const int32_t LDA,
                   const float beta,
                   float * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     float * __restrict   d_ptrA = NULL;
     float * __restrict   d_ptrC = NULL;
     float alph = alpha;
     float bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(trans==HIPBLAS_OP_N) {
         alloc_float_gpu(d_ptrA,(size_t)(LDA*k),&merr);
     }else{
         alloc_float_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrC,(size_t)(LDC*n),&merr);
     if(merr != 0) goto Error;
     if(trans==HIPBLAS_OP_N) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(LDA*k),
                          hipMemcpyHostToDevice,stream));
      }else {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(LDA*n),
                          hipMemcpyHostToDevice,stream)); 
      }
     
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasSsymm(handle,uplo,trans,n,k,&alph,d_ptrA,LDA,
                                        &bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasSsyrk executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(float)*(LDC*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  


} 


void
cuBLAS_Csyrk_iface(hipblasFillMode_t uplo,
                   hipblasOperation_t trans,
                   const int32_t n,
                   const int32_t k,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   const hipComplex beta,
                   hipComplex * __restrict C,
                   const int32_t LDC,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrC = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(trans==HIPBLAS_OP_N) {
         alloc_complex4_gpu(d_ptrA,(size_t)(LDA*k),&merr);
     }else{
         alloc_complex4_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrC,(size_t)(LDC*n),&merr);
     if(merr != 0) goto Error;
     if(trans==HIPBLAS_OP_N) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*k),
                          hipMemcpyHostToDevice,stream));
      }else {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*n),
                          hipMemcpyHostToDevice,stream)); 
      }
     
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasCsymm(handle,uplo,trans,n,k,&alph,d_ptrA,LDA,
                                        &bet,d_ptrC,LDC));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasCsyrk executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrC,d_C,sizeof(hipComplex)*(LDC*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrC));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrC) hipFree(d_ptrC);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  


} 


void
cuBLAS_Strsm_iface(hipblasSideMode_t side,
                   hipblasFillMode_t uplo,
                   hipblasOperation_t trans,
                   hipblasDiagType_t diag,
                   const int32_t m,
                   const int32_t n,
                   const float alpha,
                   const float * __restrict A,
                   const int32_t LDA,
                   float * __restrict B,
                   const int32_t LDB,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     float * __restrict   d_ptrA = NULL;
     float * __restrict   d_ptrB = NULL;
     float alph = alpha;
     float bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(side==HIPBLAS_SIDE_LEFT) {
         alloc_float_gpu(d_ptrA,(size_t)(LDA*m),&merr);
     }else{
         alloc_float_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     alloc_float_gpu(d_ptrB,(size_t)(LDB*n),&merr);
     if(merr != 0) goto Error;
     if(side==HIPBLAS_SIDE_LEFT) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(LDA*m),
                          hipMemcpyHostToDevice,stream));
      }else {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(float)*(LDA*n),
                          hipMemcpyHostToDevice,stream)); 
      }
     
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasStrsm(handle,side,uplo,trans,diag,m,n,&alph,d_ptrA,LDA,
                                        &bet,d_ptrB,LDB));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasStrsm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,d_B,sizeof(float)*(LDB*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrC) hipFree(d_ptrB);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  


} 


void
cuBLAS_Ctrsm_iface(hipblasSideMode_t side,
                   hipblasFillMode_t uplo,
                   hipblasOperation_t trans,
                   hipblasDiagType_t diag,
                   const int32_t m,
                   const int32_t n,
                   const hipComplex alpha,
                   const hipComplex * __restrict A,
                   const int32_t LDA,
                   hipComplex * __restrict B,
                   const int32_t LDB,
                   hipError_t * __restrict err,
                   int32_t * __restrict ierr,
                   uint64_t * __restrict tsc_delta) {

     if(__builtin_expect(n<=1,0)) {
       *ierr = -1;
       return;
      }
    *err    = 0;
    *ierr   = 0;
     
#if (PROFILE_HOST_TO_DEVICE) == 1
     volatile uint64_t dummy1;
     volatile uint32_t dummy2;
     volatile uint64_t tsc_start,tsc_end;
     volatile uint32_t coreid;
#endif
     hipblasHandle_t handle       = NULL;
     hipStream_t   stream       = NULL;
     hipComplex * __restrict   d_ptrA = NULL;
     hipComplex * __restrict   d_ptrB = NULL;
     hipComplex alph = alpha;
     hipComplex bet  = beta;
     cudaError_t_t status;
     int32_t merr = 0;
     GMS_CUBLAS_STAT_CHECK(hipblasCreate(&handle));
     GMS_CUDA_DEBUG_CHECK(hipStreamCreateWithFlags(&stream,hipStreamNonBlocking));
     GMS_CUBLAS_STAT_CHECK(hipblasSetStream(handle,stream));
     if(side==HIPBLAS_SIDE_LEFT) {
         alloc_complex4_gpu(d_ptrA,(size_t)(LDA*m),&merr);
     }else{
         alloc_complex4_gpu(d_ptrA,(size_t)(LDA*n),&merr);
     {
     if(merr != 0) goto Error;
     alloc_complex4_gpu(d_ptrB,(size_t)(LDB*n),&merr);
     if(merr != 0) goto Error;
     if(side==HIPBLAS_SIDE_LEFT) {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*m),
                          hipMemcpyHostToDevice,stream));
      }else {
        GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrA,A,sizeof(hipComplex)*(LDA*n),
                          hipMemcpyHostToDevice,stream)); 
      }
     
#if (PROFILE_HOST_TO_DEVICE) == 1
      dummy1    = __rdtscp(&dummy2);
      __asm__("lfence");
      tsc_start = __rdtscp(&coreid);
      __asm__("lfence");
#endif
      GMS_CUBLAS_STAT_CHECK(hipblasCtrsm(handle,side,uplo,trans,diag,m,n,&alph,d_ptrA,LDA,
                                        &bet,d_ptrB,LDB));
#if (PROFILE_HOST_TO_DEVICE) == 1
      __asm__("lfence");
      tsc_end     = __rdtscp(&coreid);
      *tsc_delta  = tsc_end-tsc_start-rdtscp_cost;
      __asm__("lfence");
      printf("hipblasCtrsm executed in: %llu reference cycles\n",*tsc_delta);
#endif  
      GMS_CUDA_DEBUG_CHECK(hipMemcpyAsync(d_ptrB,d_B,sizeof(hipComplex)*(LDB*n),
                                           hipMemcpyDeviceToHost,stream));
      GMS_CUDA_DEBUG_CHECK(hipStreamSynchronize(stream));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrA));
      GMS_CUDA_DEBUG_CHECK(hipFree(d_ptrB));
      GMS_CUBLAS_STAT_CHECK(hipblasDestroy(handle));
      GMS_CUDA_DEBUG_CHECK(hipStreamDestroy(stream));
      *ierr = 0;
      *err  = status;
      return; 
Error:   {  
           if(d_ptrA) hipFree(d_ptrA);
           if(d_ptrC) hipFree(d_ptrB);
           hipblasDestroy(handle);
           hipStreamDestroy(stream); 
           ierr = -2;
           *err = status;   
           return; 
    }  

} 
