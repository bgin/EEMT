#include "hip/hip_runtime.h"
#include "cuda/mcubes/demos/demo_utils.cuh"
#include "cuda/mcubes/vegasT.cuh"

class GENZ_6_2D {
public:
  __device__ __host__ double
  operator()(double y, double z)
  {
    if (z > .9 || y > .8)
      return 0.;
    else
      return exp(10 * z + 9 * y);
  }
};

int
main(int argc, char** argv)
{
  double epsrel = 1e-3;
  double epsrel_min = 1.e-6;
  constexpr int ndim = 2;

  double ncall = 1.0e7;
  int titer = 100;
  int itmax = 20;
  int skip = 5;
  VegasParams params(ncall, titer, itmax, skip);

  double true_value = 120489.75982636053604;

  double lows[] = {0., 0.};
  double highs[] = {1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  GENZ_6_2D integrand;

  print_mcubes_header();
  while (mcubes_time_and_call<GENZ_6_2D, ndim>(
           integrand, epsrel, true_value, "GENZ_6_2D", params, &volume) ==
           true &&
         epsrel >= epsrel_min) {
    epsrel /= 5.;
  }

  return 0;
}