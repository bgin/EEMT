#include "hip/hip_runtime.h"
#include "cuda/mcubes/demos/demo_utils.cuh"
#include "cuda/mcubes/vegasT.cuh"

class GENZ_5_8D {
public:
  __device__ __host__ double
  operator()(double x,
             double y,
             double z,
             double k,
             double m,
             double n,
             double p,
             double q)
  {
    double beta = .5;
    double t1 = -10. * fabs(x - beta) - 10. * fabs(y - beta) -
                10. * fabs(z - beta) - 10. * fabs(k - beta) -
                10. * fabs(m - beta) - 10. * fabs(n - beta) -
                10. * fabs(p - beta) - 10. * fabs(q - beta);
    return exp(t1);
  }
};

int
main(int argc, char** argv)
{
  double epsrel = 1.e-3;
  double epsrel_min = 1e-9;
  constexpr int ndim = 8;

  double ncall = 1.0e6;
  int titer = 100;
  int itmax = 20;
  int skip = 5;
  VegasParams params(ncall, titer, itmax, skip);

  double true_value = 2.425217625641885e-06;

  double lows[] = {0., 0., 0., 0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1., 1., 1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  GENZ_5_8D integrand;

  print_mcubes_header();
  std::array<double, 10> required_ncall = {
    1.e6, 1.e6, 1.e6, 1.e7, 1.e9, 1.e9, 5.e9, 8.e9, 8.e9, 8.e9};

  bool success = false;
  size_t curr_epsrel = 0;
  do {
    params.ncall = required_ncall[curr_epsrel];
    for (int run = 0; run < 100; run++) {
      success = mcubes_time_and_call<GENZ_5_8D, ndim, false, Custom_generator>(
        integrand, epsrel, true_value, "f5, 8", params, &volume);
      if (!success)
        break;
    }
    epsrel /= 5.;
    curr_epsrel++;
    if (curr_epsrel > required_ncall.size())
      break;
  } while (epsrel >= epsrel_min && success == true);
  return 0;
}
