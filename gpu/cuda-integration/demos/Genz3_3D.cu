#include "hip/hip_runtime.h"
#include "cuda/mcubes/demos/demo_utils.cuh"
#include "cuda/mcubes/vegasT.cuh"

class GENZ_3_3D {
public:
  __device__ __host__ double
  operator()(double x, double y, double z)
  {
    return pow(1 + 3 * x + 2 * y + z, -4);
  }
};

int
main(int argc, char** argv)
{
  double epsrel = 1.e-3;
  double epsrel_min = 1e-9;
  constexpr int ndim = 3;

  double ncall = 1.e7;
  int titer = 100;
  int itmax = 20;
  int skip = 0;
  VegasParams params(ncall, titer, itmax, skip);

  double true_value = 0.010846560846560846561;
  double lows[] = {0., 0., 0.};
  double highs[] = {1., 1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  GENZ_3_3D integrand;

  print_mcubes_header();
  std::array<double, 7> required_ncall =
    //{1.e7, 1.e7, 1.e7, 1.e7, 1.e7, 1.e8, 1.e9};
    {1.e7, 1.e7, 1.e7, 1.e7, 1.e7, 1.e8, 1.e9};
  bool success = false;
  size_t curr_epsrel = 0;

  do {
    params.ncall = required_ncall[curr_epsrel];
    for (int run = 0; run < 1; run++) {
      success = mcubes_time_and_call<GENZ_3_3D, ndim, false, Custom_generator>(
        integrand, epsrel, true_value, "f3, 3", params, &volume);
      if (!success)
        break;
    }
    break;
    epsrel /= 5.;
    curr_epsrel++;
    if (curr_epsrel > required_ncall.size())
      break;
  } while (epsrel >= epsrel_min && success == true);
  return 0;
}
