#include "hip/hip_runtime.h"


#include <cstdint>
// Modified by Bernard Gingold, contact: bgin@gmail.com
/*
    GLAS is licensed under the The MIT License:

Copyright (c) 2011 Hans Henrik Brandenborg Sørensen, DTU

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// Generic vector swap kernel.
//
// - Simple design for all capabilities
// - Handled by splitting matrix as
// --------------------------
// |                 |      |
// |                 |      |
// |                 |      |
// | left tiled part | rest | 
// |                 |      |
// |                 |      |
// |                 |      |
// --------------------------
// - Kernels are generic and work for all cases
//

// Version 1: copy as floats
// -----------------------------------------
// Threads per block    : BLOCKSIZE_n
// Work per thread      : WORKSIZE_n
// Shared mem per block : 0
//
template <int BLOCKSIZE_n, int WORKSIZE_m, int WORKSIZE_n>

__global__ void swap_kernel_r4(float* __restrict__ x, 
                               float* __restrict__ y, 
                               const int32_t n, 
                               const int32_t ntile,
                               const int32_t BLOCKSIZE_n, 
                               const int32_t WORKSIZE_m, 
                               const int32_t WORKSIZE_n)
{
	const int tid = threadIdx.x;
	const int idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx + tid;
	y += idx + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < WORKSIZE_n; i++ )
		{
			float temp = y[0];
			y[0] = x[0]; 
			x[0] = temp; 
			x += BLOCKSIZE_n; y += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < n - idx - tid; i += BLOCKSIZE_n)
		{
			float temp = y[i];
			y[i] = x[i]; 
			x[i] = temp; 
		}
	}
}


__global__ void swap_kernel_c4(hipComplex* __restrict__ x, 
                               hipComplex* __restrict__ y, 
                               const int32_t n, 
                               const int32_t ntile,
                               const int32_t BLOCKSIZE_n, 
                               const int32_t WORKSIZE_m, 
                               const int32_t WORKSIZE_n)
{
	const int tid = threadIdx.x;
	const int idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx + tid;
	y += idx + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < WORKSIZE_n; i++ )
		{
			hipComplex temp = y[0];
			y[0] = x[0]; 
			x[0] = temp; 
			x += BLOCKSIZE_n; y += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < n - idx - tid; i += BLOCKSIZE_n)
		{
			hipComplex temp = y[i];
			y[i] = x[i]; 
			x[i] = temp; 
		}
	}
}


__global__ void swap_kernel_r8(double* __restrict__ x, 
                               double* __restrict__ y, 
                               const int32_t n, 
                               const int32_t ntile,
                               const int32_t BLOCKSIZE_n, 
                               const int32_t WORKSIZE_m, 
                               const int32_t WORKSIZE_n)
{
	const int tid = threadIdx.x;
	const int idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx + tid;
	y += idx + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < WORKSIZE_n; i++ )
		{
			double temp = y[0];
			y[0] = x[0]; 
			x[0] = temp; 
			x += BLOCKSIZE_n; y += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < n - idx - tid; i += BLOCKSIZE_n)
		{
			double temp = y[i];
			y[i] = x[i]; 
			x[i] = temp; 
		}
	}
}

// Version 2: copy as doubles (requires WORKSIZE_n & 1 == 0)
// -----------------------------------------
// Threads per block    : BLOCKSIZE_n
// Work per thread      : WORKSIZE_n / 2
// Shared mem per block : 0
//
/*template <int BLOCKSIZE_n, int WORKSIZE_m, int WORKSIZE_n>
__launch_bounds__(BLOCKSIZE_n, MIN_BLOCKS_PER_MP)
__global__ void sswap_kernel2(float* x, float* y, int n, int ntile)
{
	const int tid = threadIdx.x;
	const int idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx;
	y += idx;
	double *xd = (double *) x + tid;
	double *yd = (double *) y + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < (WORKSIZE_n / 2); i++ )
		{
			double temp = yd[0];
			yd[0] = xd[0];
			xd[0] = temp;
			xd += BLOCKSIZE_n; yd += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < ((n - idx) >> 1) - tid; i += BLOCKSIZE_n)
		{
			double temp = yd[i];
			yd[i] = xd[i];
			xd[i] = temp;
		}
	}
	if (blockIdx.x == 0 && tid == 0 && n & 1) { float temp = y[n - 1]; y[n - 1] = x[n - 1]; x[n - 1] = temp; }
}
*/
