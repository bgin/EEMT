#include "hip/hip_runtime.h"
#include "cuda/mcubes/demos/demo_utils.cuh"
#include "cuda/mcubes/vegasT.cuh"

class GENZ_3_8D {
public:
  __device__ __host__ double
  operator()(double x,
             double y,
             double z,
             double w,
             double v,
             double u,
             double t,
             double s)
  {
    return pow(1 + 8 * s + 7 * t + 6 * u + 5 * v + 4 * w + 3 * x + 2 * y + z,
               -9);
  }
};

int
main(int argc, char** argv)
{
  double epsrel = 1e-3;
  double epsrel_min = 1e-9;
  constexpr int ndim = 8;

  double ncall = 1.0e7;
  int titer = 100;
  int itmax = 20;
  int skip = 5;
  VegasParams params(ncall, titer, itmax, skip);

  double true_value = 2.2751965817917756076e-10;

  double lows[] = {0., 0., 0., 0., 0., 0., 0., 0.};
  double highs[] = {1., 1., 1., 1., 1., 1., 1., 1.};
  quad::Volume<double, ndim> volume(lows, highs);
  GENZ_3_8D integrand;

  print_mcubes_header();

  size_t expID = 0;
  bool success = false;
  do {
    for (int run = 0; run < 1; run++) {
      success = mcubes_time_and_call<GENZ_3_8D, ndim>(
        integrand, epsrel, true_value, "f3 8D", params, &volume);
      if (!success)
        break;
    }
    break;
    epsrel /= 5.;
    expID++;
  } while (success == true && epsrel >= epsrel_min);

  return 0;
}
