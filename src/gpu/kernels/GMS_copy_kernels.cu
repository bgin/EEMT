

#include <hip/hip_runtime.h>
#include <cstdint>
// Modified by Bernard Gingold, contact: bgin@gmail.com
/*
    GLAS is licensed under the The MIT License:

Copyright (c) 2011 Hans Henrik Brandenborg Sørensen, DTU

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

// Generic vector copy kernel.
//
// - Simple design for all capabilities
// - Handled by splitting matrix as
// --------------------------
// |                 |      |
// |                 |      |
// |                 |      |
// | left tiled part | rest | 
// |                 |      |
// |                 |      |
// |                 |      |
// --------------------------
// - Kernels are generic and work for all cases
//

// Version 1: copy as floats
// -----------------------------------------
// Threads per block    : BLOCKSIZE_n
// Work per thread      : WORKSIZE_n
// Shared mem per block : 0
//

__global__ void copy_kernel_r4(const float* __restrict__ x, 
                               float* __restrict__ y, 
                               const int32_t n, 
                               const int32_t ntile,
                               const int32_t BLOCKSIZE_n,
                               const int32_t WORKSIZE_m,
                               const int32_t WORKSIZE_n )
{
	const int32_t tid = threadIdx.x;
	const int32_t idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx + tid;
	y += idx + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < WORKSIZE_n; i++ )
		{
			y[0] = x[0]; x += BLOCKSIZE_n; y += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < n - idx - tid; i += BLOCKSIZE_n)
		{
			y[i] = x[i];
		}
	}
}


__global__ void copy_kernel_r8(const double* __restrict__ x, 
                               double* __restrict__ y, 
                               const int32_t n, 
                               const int32_t ntile,
                               const int32_t BLOCKSIZE_n,
                               const int32_t WORKSIZE_m,
                               const int32_t WORKSIZE_n )
{
	const int32_t tid = threadIdx.x;
	const int32_t idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx + tid;
	y += idx + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < WORKSIZE_n; i++ )
		{
			y[0] = x[0]; x += BLOCKSIZE_n; y += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < n - idx - tid; i += BLOCKSIZE_n)
		{
			y[i] = x[i];
		}
	}
}

// Version 2: copy as doubles (requires WORKSIZE_n & 1 == 0)
// -----------------------------------------
// Threads per block    : BLOCKSIZE_n
// Work per thread      : WORKSIZE_n / 2
// Shared mem per block : 0
//
/*template <int BLOCKSIZE_n, int WORKSIZE_m, int WORKSIZE_n>
__launch_bounds__(BLOCKSIZE_n, MIN_BLOCKS_PER_MP)
__global__ void scopy_kernel2(const float* x, float* y, int n, int ntile)
{
	const int tid = threadIdx.x;
	const int idx = blockIdx.x * BLOCKSIZE_n * WORKSIZE_n;
	x += idx;
	y += idx;
	const double *xd = (double *) x + tid;
	double *yd = (double *) y + tid;
	if (idx != ntile)
	{
        #pragma unroll
		for (int i = 0; i < (WORKSIZE_n / 2); i++ )
		{
			yd[0] = xd[0]; xd += BLOCKSIZE_n; yd += BLOCKSIZE_n;
		}
	}
	else
	{
        #pragma unroll 4
		for (int i = 0; i < ((n - idx) >> 1) - tid; i += BLOCKSIZE_n)
		{
			yd[i] = xd[i];
		}
	}
	if (blockIdx.x == 0 && tid == 0 && n & 1) y[n - 1] = x[n - 1];
}
*/
