#include "hip/hip_runtime.h"




#include "GMS_radar_jamming_kernels.cuh"
#include "GMS_radar_jamming_common.cuh"


static __device__
float therm_noise_range(   const float Frdr,
                           const float Kth,
                           const float rho,
                           const float tf,
                           const float tr,
	                   const float th,
		           const float d_Pt,
		           const float gamm,
			   const float d_w,
			   const float d_h,
			   const float Ln,
			   const float d_Ts,
			   const float sig,
			   const float F,
			   const float Fp,
		           const float Flens,
			   const float Dx,
			   const float d_Lt,
			   const float d_La) {

        const float dc  = duty_cycle(rho,tr);
        const float Pav = radar_avg_pow(d_Pt,dc);
        const float ag  = radar_ant_gain(azimuth_bw(Kth,gamm,
                                                    d_h),
                                         elevation_bw(Kth,gamm,
                                                    d_w),Ln);
        const float N0  = noise_density(d_Ts);
        const float den = 1984.4017075391884912304967f*N0*Dx*d_Lt*d_La;
        const float t1  = gamm*gamm;
        const float t2  = Pav*tf;
        const float t3  = ag*ag;
        const float t4  = sig*Frdr*Frdr*Fp*Fp;
        const float t5  = F*F*F*F*Flens*Flens;
        const float num = t1*t2*t3*t4*t5;
        const float rat = num/den;
        return (powf(rat,0.25f));
}


__global__ void
therm_noise_range_kernel1(  const float Frdr,
                            const float Kth,
                            const float rho,
                            const float tf,
                            const float tr,
	                    const float th,
		            const float * __restrict d_Pt,
		            const float gamm,
			    const float * __restrict d_w,
			    const float * __restrict d_h,
			    const float Ln,
			    const float * __restrict d_Ts,
			    const float sig,
			    const float F,
			    const float Fp,
		            const float Flens,
			    const float Dx,
			    const float * __restrict d_Lt,
			    const float * __restrict d_La,
			    float * __restrict d_Rm,
			    const uint32_t n_threads) {

     uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
     if(tid < n_threads) {
        const float dc  = duty_cycle(rho,tr);
        const float Pav = radar_avg_pow(d_Pt[tid],dc);
        const float ag  = radar_ant_gain(azimuth_bw(Kth,gamm,
                                                    d_h[tid]),
                                         elevation_bw(Kth,gamm,
                                                    d_w[tid]),Ln);
        const float N0  = noise_density(d_Ts[tid]);
        const float den = 1984.4017075391884912304967f*N0*Dx*d_Lt[tid]*d_La[tid];
        const float t1  = gamm*gamm;
        const float t2  = Pav*tf;
        const float t3  = ag*ag;
        const float t4  = sig*Frdr*Frdr*Fp*Fp;
        const float t5  = F*F*F*F*Flens*Flens;
        const float num = t1*t2*t3*t4*t5;
        const float rat = num/den;
        Rm[tid]         = powf(rat,0.25f);
     }
}


void therm_noise_range1_cuda( const float Frdr,
                              const float Kth,
                              const float rho,
                              const float tf,
                              const float tr,
	                      const float th,
		              const float * __restrict d_Pt,
		              const float gamm,
			      const float * __restrict d_w,
			      const float * __restrict d_h,
			      const float Ln,
			      const float * __restrict d_Ts,
			      const float sig,
			      const float F,
			      const float Fp,
		              const float Flens,
			      const float Dx,
			      const float * __restrict d_Lt,
			      const float * __restrict d_La,
			      float * __restrict d_Rm,
			      const uint32_t n_threads) {

         uint threadsBlock = 32;
         uint blocksGrid  = (n_threads + threadsBlock - 1) / threadsBlock;
         therm_noise_range_kernel1<<<blocksGrid,threadsBlock>>>( Frdr,Kth,rho,tf,tr,th,d_Pt,gamm,
                                                                 d_w,d_h,Ln,d_Ts,sig,F,Fp,Flens,Dx,
                                                                 d_Lt,d_La,d_Rm,n_threads);
}

__global__ void
therm_noise_range_kernel2(  const float Frdr,
                            const float Kth,
                            const float rho,
                           const float tf,
                           const float tr,
	                   const float th,
		           const float * __restrict d_Pt,
		           const float gamm,
			   const float * __restrict d_w,
			   const float * __restrict d_h,
			   const float Ln,
			   const float * __restrict d_Ts,
			   const float sig,
			   const float F,
			   const float Fp,
		           const float Flens,
			   const float Dx,
			   const float * __restrict d_Lt,
			   const float * __restrict d_La,
			   float * __restrict d_Rm,
			   const uint32_t n) {

     uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
     uint32_t stride = blockDim.x*gridDim.x;
     for(uint32_t i = tid; i < n; i += stride) {
         const float dc  = duty_cycle(rho,tr);
         const float Pav = radar_avg_pow(d_Pt[i],dc);
         const float ag  = radar_ant_gain(azimuth_bw(Kth,gamm,
                                                    d_h[i]),
                                         elevation_bw(Kth,gamm,
                                                    d_w[i]),Ln);
         const float N0  = noise_density(d_Ts[i]);
         const float den = 1984.4017075391884912304967f*N0*Dx*d_Lt[i]*d_La[i];
         const float t1  = gamm*gamm;
         const float t2  = Pav*tf;
         const float t3  = ag*ag;
         const float t4  = sig*Frdr*Frdr*Fp*Fp;
         const float t5  = F*F*F*F*Flens*Flens;
         const float num = t1*t2*t3*t4*t5;
         const float rat = num/den;
         Rm[i]         = powf(rat,0.25f);
     }
}

void therm_noise_range2_cuda( const float Frdr,
                              const float Kth,
                              const float rho,
                              const float tf,
                              const float tr,
	                      const float th,
		              const float * __restrict d_Pt,
		              const float gamm,
			      const float * __restrict d_w,
			      const float * __restrict d_h,
			      const float Ln,
			      const float * __restrict d_Ts,
			      const float sig,
			      const float F,
			      const float Fp,
		              const float Flens,
			      const float Dx,
			      const float * __restrict d_Lt,
			      const float * __restrict d_La,
			      float * __restrict d_Rm,
			      const uint32_t n) {

         uint threadsBlock = 256;
         uint blocksGrid  = (n + threadsBlock - 1) / threadsBlock;
         therm_noise_range_kernel2<<<blocksGrid,threadsBlock>>>( Frdr,Kth,rho,tf,tr,th,d_Pt,gamm,
                                                                 d_w,d_h,Ln,d_Ts,sig,F,Fp,Flens,Dx,
                                                                 d_Lt,d_La,d_Rm,n);


static __device__
float tropo_range_loss(     const float Frdr,
                            const float Kth,
                            const float rho,
                            const float tf,
                            const float tr,
	                    const float th,
		            const float Pt,
			    const float Rmj,
			    const float gamm,
			    const float w,
			    const float h,
			    const float Ln,
			    const float Ts,
			    const float sig,
			    const float F,
			    const float Fp,
		            const float Flens,
			    const float Dx,
			    const float Lt,
			    const float La,
			    float Rm) {

           
           const float Rm = therm_noise_range(Frdr,Kth,rho,tf,tr,th,
                                              Pt,Rmj,gamm,
                                              w,h,Ln,Ts,
                                              sig,F,FP,Flens,Dx,Lt,La,Rm);
           return (La*(Rmj/Rm));                              
} 


__global__
void jammer_req_temp_kernel1( const float Frdr,
                              const float Kth,
                              const float rho,
                              const float tf,
                              const float tr,
                              const float th,
                              const float * __restrict__ d_Pt,
                              const float gamm,
                              const float * __restrict__ d_w,
                              const float * __restrict__ d_h,
                              const float Ln,
                              const float * __restrict__ d_Ts,
                              const float sig,
                              const float F,
                              const float Fp,
                              const float Flens,
                              const float Dx,
                              const float * __restrict__ d_Lt,
                              const float * __restrict__ d_Rm,
                              const float Rmj,
                              const float La,
                              const float Flen,
                              float * __restrict__ rt,
                              const uint32_t n_threads) {

    uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
    
    if(tid < n_threads) {
       Ts               = d_Ts[tid];
       const float Rm   = therm_noise_range(Frdr,Kth,rho,tf,tr,th,
                                            d_Pt[tid],Rmj,gamm,
                                            d_w[tid],d_h[tid],Ln,d_Ts[tid],
                                            sig,F,FP,Flens,Dx,d_Lt[tid],La,d_Rm[tid]);
                                            
       const float La1  = tropo_range_loss(Frdr,Kth,rho,tf,tr,th,d_Pt[tid],
                                           Rmj,gamm,d_w[tid],d_h[tid],Ln,d_Ts[tid],
                                           sig,F,Fp,Flens,Dx,d_Lt[tid],La);
       const float Fln1 = sqrtf(Flen);
       const float lrat = La/La1;
       const float mrat = Flen/Fln1;
       const float rrat = Rm/Rmj;
       const float rrat4= (rrat*rrat*rrat*rrat)-1.0f;
       rt[tid]          = Ts*lrat*(mrat*mrat)*rrat4;
    }
}

__global__
void jammer_req_temp_kernel2( const float Frdr,
                              const float Kth,
                              const float rho,
                              const float tf,
                              const float tr,
                              const float th,
                              const float * __restrict__ d_Pt,
                              const float gamm,
                              const float * __restrict__ d_w,
                              const float * __restrict__ d_h,
                              const float Ln,
                              const float * __restrict__ d_Ts,
                              const float sig,
                              const float F,
                              const float Fp,
                              const float Flens,
                              const float Dx,
                              const float * __restrict__ d_Lt,
                              const float * __restrict__ d_Rm,
                              const float Rmj,
                              const float La,
                              const float Flen,
                              float * __restrict__ rt,
                              const uint32_t n) {

   
    uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
    uint32_t stride = blockDim.x*gridDim.x;
    for(uint32_t i = tid; i < n; i += stride) {
       Ts               = d_Ts[tid];
       const float Rm   = therm_noise_range(Frdr,Kth,rho,tf,tr,th,
                                            d_Pt[i],Rmj,gamm,
                                            d_w[i],d_h[i],Ln,d_Ts[i],
                                            sig,F,FP,Flens,Dx,d_Lt[i],La,d_Rm[i]);
                                            
       const float La1  = tropo_range_loss(Frdr,Kth,rho,tf,tr,th,d_Pt[i],
                                           Rmj,gamm,d_w[i],d_h[i],Ln,d_Ts[i],
                                           sig,F,Fp,Flens,Dx,d_Lt[i],La);
       const float Fln1 = sqrtf(Flen);
       const float lrat = La/La1;
       const float mrat = Flen/Fln1;
       const float rrat = Rm/Rmj;
       const float rrat4= (rrat*rrat*rrat*rrat)-1.0f;
       rt[i]          = Ts*lrat*(mrat*mrat)*rrat4;
    }
}

	 
void jammer_req_temp_cuda(    const float Frdr,
                              const float Kth,
                              const float rho,
                              const float tf,
                              const float tr,
                              const float th,
                              const float * __restrict__ d_Pt,
                              const float gamm,
                              const float * __restrict__ d_w,
                              const float * __restrict__ d_h,
                              const float Ln,
                              const float * __restrict__ d_Ts,
                              const float sig,
                              const float F,
                              const float Fp,
                              const float Flens,
                              const float Dx,
                              const float * __restrict__ d_Lt,
                              const float * __restrict__ d_Rm,
                              const float Rmj,
                              const float La,
                              const float Flen,
                              float * __restrict__ rt,
                              const uint32_t n_threads,
                              const uint32_t type,
                              const uint32_t n) {

     if(type==1U) {
         uint threadsBlock = 32;
         uint blocksGrid  = (n_threads + threadsBlock - 1) / threadsBlock;
         jammer_req_temp_kernel1<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,d_Pt,
                                                              gamm,d_w,d_h,Ln,d_Ts,sig,F,
                                                              Fp,Flens,Dx,d_Lt,d_Rm,Rmj,
                                                              La,Flen,rt,n_threads);
     }
     else if(type==2U) {
         uint threadsBlock = 256;
         uint blocksGrid  = (n + threadsBlock - 1) / threadsBlock;
         jammer_req_temp_kernel1<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,d_Pt,
                                                              gamm,d_w,d_h,Ln,d_Ts,sig,F,
                                                              Fp,Flens,Dx,d_Lt,d_Rm,Rmj,
                                                              La,Flen,rt,n); 
     }
}
			    

__global__ void
tropo_range_loss_kernel1(   const float Frdr,
                            const float Kth,
                            const float rho,
                            const float tf,
                            const float tr,
	                    const float th,
		            const float * __restrict d_Pt,
			    const float * __restrict d_Rmj,
			    const float gamm,
			    const float * __restrict d_w,
			    const float * __restrict d_h,
			    const float Ln,
			    const float * __restrict d_Ts,
			    const float sig,
			    const float F,
			    const float Fp,
		            const float Flens,
			    const float Dx,
			    const float * __restrict d_Lt,
			    const float * __restrict d_La,
			    float * __restrict d_Rm,
			    float * __restrict d_La1,
			    const uint n_threads,
                            const int32_t type,
                            const uint32_t n) {

        
     /* if(type==1) {
         uint threadsBlock = 32;
         uint blocksGrid  = (n_threads + threadsBlock - 1) / threadsBlock;
	 therm_noise_range_kernel1<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,
	                                                      d_Pt,gamm,d_w,d_h,Ln,
							      d_Ts,sig,F,Fp,Flens,
							      Dx,d_Lt,d_La,d_Rm,n_threads);
      }
      else if(type==2) {
         uint threadsBlock = 256;
         uint blocksGrid  = (n + threadsBlock - 1) / threadsBlock;
	 therm_noise_range_kernel2<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,
	                                                      d_Pt,gamm.d_w,d_h,Ln,
							      d_Ts,sig,F,Fp,Flens,
							      Dx,d_Lt,d_La,d_Rm,n);
      }*/
	uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
        if(tid < n_threads) {
           const float La = d_La[tid];
	   const float Rmj= d_Rmj[tid];
           const float Rm = therm_noise_range(Frdr,Kth,rho,tf,tr,th,
                                              d_Pt[tid],Rmj,gamm,
                                              d_w[tid],d_h[tid],Ln,d_Ts[tid],
                                              sig,F,FP,Flens,Dx,d_Lt[tid],La,
                                              d_Rm[tid]);
           
	   d_La1[tid]     = La*(Rmj/Rm);
        }
}


__global__ void
tropo_range_loss_kernel2(   const float Frdr,
                            const float Kth,
                            const float rho,
                            const float tf,
                            const float tr,
	                    const float th,
		            const float * __restrict d_Pt,
			    const float * __restrict d_Rmj,
			    const float gamm,
			    const float * __restrict d_w,
			    const float * __restrict d_h,
			    const float Ln,
			    const float * __restrict d_Ts,
			    const float sig,
			    const float F,
			    const float Fp,
		            const float Flens,
			    const float Dx,
			    const float * __restrict d_Lt,
			    const float * __restrict d_La,
			    float * __restrict d_Rm,
			    float * __restrict d_La1,
			    const uint n_threads,
                            const int32_t type,
                            const uint32_t n) {

        
      /*if(type==1) {
         uint threadsBlock = 32;
         uint blocksGrid  = (n_threads + threadsBlock - 1) / threadsBlock;
	 therm_noise_range_kernel1<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,
	                                                      d_Pt,gamm.d_w,d_h,Ln,
							      d_Ts,sig,F,Fp,Flens,
							      Dx,d_Lt,d_La,d_Rm,n_threads);
      }
      else if(type==2) {
         uint threadsBlock = 256;
         uint blocksGrid  = (n + threadsBlock - 1) / threadsBlock;
	 therm_noise_range_kernel2<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,
	                                                      d_Pt,gamm.d_w,d_h,Ln,
							      d_Ts,sig,F,Fp,Flens,
							      Dx,d_Lt,d_La,d_Rm,n);
      }*/
      uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
      uint32_t stride = blockDim.x*gridDim.x;
      for(uint32_t i = tid; i < n; i += stride) {
           const float La = d_La[i];
	   const float Rmj= d_Rmj[i];
           const float Rm = therm_noise_range(Frdr,Kth,rho,tf,tr,th,
                                              d_Pt[i],Rmj,gamm,
                                              d_w[tid],d_h[i],Ln,d_Ts[i],
                                              sig,F,FP,Flens,Dx,d_Lt[i],La,
                                              d_Rm[i]);
           
	   d_La1[i]     = La*(Rmj/Rm);
        }
}




 void
 therm_noise_range_cuda(           const float Frdr,
                                   const float Kth,
                                   const float rho,
                                   const float tf,
                                   const float tr,
				   const float th,
				   const float * __restrict d_Pt,
				   const float gamm,
				   const float * __restrict d_w,
				   const float * __restrict d_h,
				   const float Ln,
				   const float * __restrict d_Ts,
				   const float sig,
				   const float F,
				   const float Fp,
				   const float Flens,
				   const float Dx,
				   const float * __restrict d_Lt,
				   const float * __restrict d_La,
				   float * __restrict d_Rm,
				   const uint32_t n_threads,
                                   const uint32_t type,
                                   const uint32_t n) {
	  
          if(type==1) {			  
             uint32_t threadsBlock = 32;
             uint32_t blocksGrid  = (n_threads + threadsBlock - 1) / threadsBlock;
             therm_noise_range_kernel1<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,
	                                                            d_Pt,gamm.d_w,d_h,Ln,
							            d_Ts,sig,F,Fp,Flens,
							            Dx,d_Lt,d_La,d_Rm,n_threads);
           }
           else if(type==2) {
             uint32_t threadsBlock = 256;
             uint32_t blocksGrid  = (n_threads + threadsBlock - 1) / threadsBlock;
             therm_noise_range_kernel2<<<blocksGrid,threadsBlock>>>(Frdr,Kth,rho,tf,tr,th,
	                                                            d_Pt,gamm.d_w,d_h,Ln,
							            d_Ts,sig,F,Fp,Flens,
							            Dx,d_Lt,d_La,d_Rm,n);
           }


}

 void
 tropo_range_loss_cuda(            const float Frdr,
                                   const float Kth,
                                   const float rho,
                                   const float tf
                                   const float tr,
				   const float th,
				   const float * __restrict d_Pt,
				   const float * __restrict d_Rmj,
				   const float gamm,
				   const float * __restrict d_w,
				   const float * __restrict d_h,
				   const float Ln,
				   const float * __restrict d_Ts,
				   const float sig,
				   const float F,
				   const float Fp,
				   const float Flens,
				   const float Dx,
				   const float * __restrict d_Lt,
				   const float * __restrict d_La,
				   float * __restrict d_Rm,
				   float * __restrict d_La1,
				   const uint32_t n_threads,
                                   const uint32_t type,
                                   const uint32_t n) {
         
         if(type==1) {
            uint32_t threadsBlock = 32;
            uint32_t blocksGrid   = (n_threads + threadsBlock - 1) / threadsBlock;
	    tropo_range_loss_kernel1<<<blocksGrid,threadsBlock>>>( Frdr,Kth,rho,tf,tr,d_Pt,d_Rmj,
	                                                           gamm,d_w,d_h,Ln,d_Ts,sig,F,Fp,
								   Flens,Dx,d_Lt,d_La,d_Rm,d_La1,
                                                                   n_threads,type,n);
         }
         else if(type==2) {
            uint32_t threadsBlock = 256;
            uint32_t blocksGrid   = (n + threadsBlock - 1) / threadsBlock;
	    tropo_range_loss_kernel2<<<blocksGrid,threadsBlock>>>( Frdr,Kth,rho,tf,tr,d_Pt,d_Rmj,
	                                                           gamm,d_w,d_h,Ln,d_Ts,sig,F,Fp,
								   Flens,Dx,d_Lt,d_La,d_Rm,d_La1,
                                                                   n_threads,type,n);
         }
}

 // Effective radiated power of Jammer (W)
/*
                            float sig;  //m, RSC of target
			    float Pj;   //W, jammer power
			    float Gj;   //dB, jammer antenna gain
			    float Qj;   //dB, jammer noise quality
			    float Flenj;//dB, jammer lens factor
			    float Rj;   //km, jammer range
			    float Bj;   //Mhz,jammer noise BW
			    float Ltj;  //dB, jammer transmit loss
			    float Fpj;  //dB, jammer polarization
			    float Rmj;  //km, jammer screening range
			    float Fj;   //dB, jammer pattern factor of propagation
			    float Laj;  //dB, jammer troposhperic loss
*/

__global__
void jammer_erp_kernel1(const float * __restrict__ Pj,
                        const float Gj,
                        const float Ltj,
                        float * __restrict__ erp,
                        const uint32_t n_threads) {

      uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
      if(tid < n_threads) {
         const float xPj = Pj[tid];
         erp[tid] = (xPj*Gj)/Ltj;
      }
}


__global__
void jammer_erp_kernel2(const float * __restrict__ Pj,
                        const float Gj,
                        const float Ltj,
                        float * __restrict__ erp,
                        const uint32_t n) {

     uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
     uint32_t stride = blockDim.x*gridDim.x;
     for(uint32_t i = tid; i < n; i += stride) {
          const float xPj = Pj[i];
          erp[i] = (xPj*Gj)/Ltj;
     }
}


void jammer_erp_cuda(const float * __restrict__ Pj,
                     const float Gj,
                     const float Ltj,
                     float * __restrict__ erp,
                     const uint32_t n_threads,
                     const uint32_t type,
                     const uint32_t n) {
    
     if(type==1) {
         uint32_t threadsBlock = 32;
         uint32_t blocksGrid   = (n_threads + threadsBlock - 1) / threadsBlock;
         jammer_erp_kernel1<<<blocksGrid,threadsBlock>>>(Pj,Gj,Ltj,erp,n_threads);
     }
     else if(type==2) {
         uint32_t threadsBlock = 256;
         uint32_t blocksGrid   = (n + threadsBlock - 1) / threadsBlock;
         jammer_erp_kernel2<<<blocksGrid,threadsBlock>>>(Pj,Gj,Ltj,erp,n);
     }
}


// Effective radiated Jammer noise power (W)
                        
__global__
void jammer_ernp_kernel1(const float * __restrict__ Pj,
                         const float Qj,
                         const float Gj,
                         const float Fpj,
                         const float Ltj,
                         float * __restrict__ ernp,
                         const uint32_t n_threads) {

      uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
      const float xQj = Qj;
      const float xGj = Gj;
      const float xFpj= Fpj;
      const float xLtj= Ltj;
      if(tid < n_threads) {
          const float xPj = Pj[tid];
          const float xFpj2 = xFpj*xFpj;
          ernp[tid] = (xQj*xPj*xGj*xFpj2)/xLtj;
      }
}


__global__
void jammer_ernp_kernel2(const float * __restrict__ Pj,
                         const float Qj,
                         const float Gj,
                         const float Fpj,
                         const float Ltj,
                         float * __restrict__ ernp,
                         const uint32_t n) {

     uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
     uint32_t stride = blockDim.x*gridDim.x; 
     const float xQj = Qj;
     const float xGj = Gj;
     const float xFpj= Fpj;
     const float xLtj= Ltj;
     for(uint32_t i = tid; i < n; i += stride) {
          const float xPj = Pj[i];
          const float xFpj2 = xFpj*xFpj;
          ernp[i] = (xQj*xPj*xGj*xFpj2)/xLtj;
     }
} 


void jammer_ernp_cuda(const float * __restrict__ Pj,
                      const float Qj,
                      const float Gj,
                      const float Fpj,
                      const float Ltj,
                      float * __restrict__ ernp,
                      const uint32_t n_threads,
                      const uint32_t type,
                      const uint32_t n) {

     if(type==1) {
         uint32_t threadsBlock = 32;
         uint32_t blocksGrid   = (n_threads + threadsBlock - 1) / threadsBlock;
         jammer_ernp_kernel1<<<blocksGrid,threadsBlock>>>(Pj,Qj,Gj,Fpj,Ltj,ernp,n_threads);
     }
     else if(type==2) {
         uint32_t threadsBlock = 256;
         uint32_t blocksGrid   = (n + threadsBlock - 1) / threadsBlock;
         jammer_ernp_kerne2<<<blocksGrid,threadsBlock>>>(Pj,Qj,Gj,Fpj,Ltj,ernp,n);
     }
}

 // Jamming spectral density (W/Hz)

__global__
void jammer_spectr_dens_kernel1(const float * __restrict__ gamma,
                                const float Kth,
                                const float Ln,
                                const float h,
                                const float w,
                                const float Qj,
                                const float * __restrict__ Pj,
                                const float Gj,
                                const float Fpj,
                                const float Flnsj,
                                const float Fj,
                                const float Rj,
                                const float Bj,
                                const float Ltj,
                                const float Laj,
                                float * __restrict__ sd,
                                const uint32_t n_threads) {
          
        const float j0   = 0.0f;
        const float PI42 =  157.9136704174297379013522f;  
        uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
        if(tid < n_threads) {
           const float xgamma = gamma[tid];
           const float xPj    = Pj[tid];
           const float xgm2   = xgamma*xgamma;
           const float xGr    = radar_ant_gain(azimuth_bw(Kth,xgamma,h),
                                               elevation_bw(Kth,xgamma,w),Ln);
           const float r0     = Rj*Bj*Ltj*Laj;
           const float t0     = Qj*xPj*Gj*Gr;
           const float t1     = xgm2*Fpj*Fpj*Flnsj*Flnsj;
           const float t2     = t0*t1*Fj*Fj;
           j0                 = t2/(PI42*r0);
           sd[tid]            = j0;
        } 
}


__global__
void jammer_spectr_dens_kernel2(const float * __restrict__ gamma,
                                const float Kth,
                                const float Ln,
                                const float h,
                                const float w,
                                const float Qj,
                                const float * __restrict__ Pj,
                                const float Gj,
                                const float Fpj,
                                const float Flnsj,
                                const float Fj,
                                const float Rj,
                                const float Bj,
                                const float Ltj,
                                const float Laj,
                                float * __restrict__ sd,
                                const uint32_t n) {

      const float j0   = 0.0f;
      const float PI42 =  157.9136704174297379013522f;
      uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
      uint32_t stride = blockDim.x*gridDim.x; 
      for(uint32_t i = tid; i < n; i += stride) {
           const float xgamma = gamma[i];
           const float xPj    = Pj[i];
           const float xgm2   = xgamma*xgamma;
           const float xGr    = radar_ant_gain(azimuth_bw(Kth,xgamma,h),
                                               elevation_bw(Kth,xgamma,w),Ln);
           const float r0     = Rj*Bj*Ltj*Laj;
           const float t0     = Qj*xPj*Gj*Gr;
           const float t1     = xgm2*Fpj*Fpj*Flnsj*Flnsj;
           const float t2     = t0*t1*Fj*Fj;
           j0                 = t2/(PI42*r0);
           sd[i]            = j0;
      }  
}


void jammer_spectr_dens_cuda(   const float * __restrict__ gamma,
                                const float Kth,
                                const float Ln,
                                const float h,
                                const float w,
                                const float Qj,
                                const float * __restrict__ Pj,
                                const float Gj,
                                const float Fpj,
                                const float Flnsj,
                                const float Fj,
                                const float Rj,
                                const float Bj,
                                const float Ltj,
                                const float Laj,
                                float * __restrict__ sd,
                                const uint32_t n_threads,
                                const uint32_t type,
                                const uint32_t n) {

     if(type==1) {
         uint32_t threadsBlock = 32;
         uint32_t blocksGrid   = (n_threads + threadsBlock - 1) / threadsBlock;
         jammer_spectr_dens_kernel1<<<blocksGrid,threadsBlock>>>(gamma,Kth,Ln,h,w,Qj,Pj,
                                                                 Gj,Fpj,Flnsj,Fj,Rj,Bj,
                                                                 Ltj,Laj,sd,n_threads);
     }
     else if(type==2) {
         uint32_t threadsBlock = 256;
         uint32_t blocksGrid   = (n + threadsBlock - 1) / threadsBlock;
         jammer_spectr_dens_kernel2<<<blocksGrid,threadsBlock>>>(gamma,Kth,Ln,h,w,Qj,Pj,
                                                                 Gj,Fpj,Flnsj,Fj,Rj,Bj,
                                                                 Ltj,Laj,sd,n);
     }
}

// Available jamming temperature single jammer scenario
// Remark: the implementation is identical to function coded above.

__global__
void single_jammer_temp_kernel1(const float * __restrict__ gamma,
                                const float Kth,
                                const float Ln,
                                const float h,
                                const float w,
                                const float Qj,
                                const float * __restrict__ Pj,
                                const float Gj,
                                const float Fpj,
                                const float Flnsj,
                                const float Fj,
                                const float Rj,
                                const float Bj,
                                const float Ltj,
                                const float Laj,
                                float * __restrict__ sd,
                                const uint32_t n_threads) {
          
        const float jt1   = 0.0f;
        const float PI42 =  157.9136704174297379013522f;  
        uint32_t tid = blockDim.x*blockIdx.x+threadIdx.x;
        if(tid < n_threads) {
           const float xgamma = gamma[tid];
           const float xPj    = Pj[tid];
           const float xgm2   = xgamma*xgamma;
           const float xGr    = radar_ant_gain(azimuth_bw(Kth,xgamma,h),
                                               elevation_bw(Kth,xgamma,w),Ln);
           const float r0     = Rj*k_B*Bj*Ltj*Laj;
           const float t0     = Qj*xPj*Gj*Gr;
           const float t1     = xgm2*Fpj*Fpj*Flnsj*Flnsj;
           const float t2     = t0*t1*Fj*Fj;
           jt1                = t2/(PI42*r0);
           sd[tid]            = jt1;
        } 
}


__global__
void single_jammer_temp_kernel2(const float * __restrict__ gamma,
                                const float Kth,
                                const float Ln,
                                const float h,
                                const float w,
                                const float Qj,
                                const float * __restrict__ Pj,
                                const float Gj,
                                const float Fpj,
                                const float Flnsj,
                                const float Fj,
                                const float Rj,
                                const float Bj,
                                const float Ltj,
                                const float Laj,
                                float * __restrict__ sd,
                                const uint32_t n) {

      const float jt1  = 0.0f;
      const float PI42 =  157.9136704174297379013522f;
      uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
      uint32_t stride = blockDim.x*gridDim.x; 
      for(uint32_t i = tid; i < n; i += stride) {
           const float xgamma = gamma[i];
           const float xPj    = Pj[i];
           const float xgm2   = xgamma*xgamma;
           const float xGr    = radar_ant_gain(azimuth_bw(Kth,xgamma,h),
                                               elevation_bw(Kth,xgamma,w),Ln);
           const float r0     = Rj*k_B*Bj*Ltj*Laj;
           const float t0     = Qj*xPj*Gj*Gr;
           const float t1     = xgm2*Fpj*Fpj*Flnsj*Flnsj;
           const float t2     = t0*t1*Fj*Fj;
           jt1                = t2/(PI42*r0);
           sd[i]              = jt1;
      }  
}


void single_jammer_temp_cuda(   const float * __restrict__ gamma,
                                const float Kth,
                                const float Ln,
                                const float h,
                                const float w,
                                const float Qj,
                                const float * __restrict__ Pj,
                                const float Gj,
                                const float Fpj,
                                const float Flnsj,
                                const float Fj,
                                const float Rj,
                                const float Bj,
                                const float Ltj,
                                const float Laj,
                                float * __restrict__ sd,
                                const uint32_t n_threads,
                                const uint32_t type,
                                const uint32_t n) {

     if(type==1) {
         uint32_t threadsBlock = 32;
         uint32_t blocksGrid   = (n_threads + threadsBlock - 1) / threadsBlock;
         single_jammer_temp_kernel1<<<blocksGrid,threadsBlock>>>(gamma,Kth,Ln,h,w,Qj,Pj,
                                                                 Gj,Fpj,Flnsj,Fj,Rj,Bj,
                                                                 Ltj,Laj,sd,n_threads);
     }
     else if(type==2) {
         uint32_t threadsBlock = 256;
         uint32_t blocksGrid   = (n + threadsBlock - 1) / threadsBlock;
         single_jammer_temp_kernel2<<<blocksGrid,threadsBlock>>>(gamma,Kth,Ln,h,w,Qj,Pj,
                                                                 Gj,Fpj,Flnsj,Fj,Rj,Bj,
                                                                 Ltj,Laj,sd,n);
     }
}


//// Number of jammers range (km)
__global__
void n_jammers_range_kernel1(const float Kth,
                             const float gamma,
                             const float * __restrict__ h,
                             const float * __restrict__ w,
                             const float Ln,
                             const float rho,
                             const float tf,
                             const float tr,
                             const float * __restrict__ Pt,
                             const float Frdr,
                             const float Fp,
                             const float F,
                             const float Flen,
                             const float * __restrict__ sig,
                             const float Ts,
                             const float Dx,
                             const float Lt,
                             const float La,
                             float * __restrict Rnj,
                             const uint32_t n_threads) {
       
       const float c0 = 1984.4017075391884912304842f
       uint32_t tid   = blockDim.x*blockIdx.x+threadIdx.x;
       if(tid < n_threads) {
          const float xh     = h[tid];
          const float xw     = w[tid];
          const float xPt    = Pt[tid];
          const float xsig   = sig[tid];
          const float gamma2 = gamma*gamma;
          const float Gr     = radar_ant_gain(azimuth_bw(Kth,gamma2,xh),
                                              elevation_bw(Kth,gamma2,xw),Ln);
          const float Gr2    = Gr*Gr;
          const float dc     = duty_cycle(rho,tr);
          const float pav    = radar_avg_power(xPt,dc);
          const float N0     = noise_density(Ts);
          const float F4     = F*F*F*F;
          const float num    = pav*tf*Gr2*gamma*xsig*F4*Fp;
          const float den    = c0*k_B*Ts*Dx*Lt*La;
          const float ratio  = num/den;
          Rnj[tid]           = powf(ratio,0.25f);
       } 
}

__global__
void n_jammers_range_kernel2(const float Kth,
                             const float gamma,
                             const float * __restrict__ h,
                             const float * __restrict__ w,
                             const float Ln,
                             const float rho,
                             const float tf,
                             const float tr,
                             const float * __restrict__ Pt,
                             const float Frdr,
                             const float Fp,
                             const float F,
                             const float Flen,
                             const float * __restrict__ sig,
                             const float Ts,
                             const float Dx,
                             const float Lt,
                             const float La,
                             float * __restrict Rnj,
                             const uint32_t n) {
       
       const float c0 = 1984.4017075391884912304842f
       uint32_t tid    = blockIdx.x*blockDim.x+threadIdx.x;
       uint32_t stride = blockDim.x*gridDim.x; 
       for(uint32_t i = tid; i < n; i += stride) {   
           const float xh     = h[i];
           const float xw     = w[i];
           const float xPt    = Pt[i];
           const float xsig   = sig[i];
           const float gamma2 = gamma*gamma;
           const float Gr     = radar_ant_gain(azimuth_bw(Kth,gamma2,xh),
                                              elevation_bw(Kth,gamma2,xw),Ln);
           const float Gr2    = Gr*Gr;
           const float dc     = duty_cycle(rho,tr);
           const float pav    = radar_avg_power(xPt,dc);
           const float N0     = noise_density(Ts);
           const float F4     = F*F*F*F;
           const float num    = pav*tf*Gr2*gamma*xsig*F4*Fp;
           const float den    = c0*k_B*Ts*Dx*Lt*La;
           const float ratio  = num/den;
           Rnj[i]           = powf(ratio,0.25f);
       } 
} 


void n_jammers_range_cuda(   const float Kth,
                             const float gamma,
                             const float * __restrict__ h,
                             const float * __restrict__ w,
                             const float Ln,
                             const float rho,
                             const float tf,
                             const float tr,
                             const float * __restrict__ Pt,
                             const float Frdr,
                             const float Fp,
                             const float F,
                             const float Flen,
                             const float * __restrict__ sig,
                             const float Ts,
                             const float Dx,
                             const float Lt,
                             const float La,
                             float * __restrict Rnj,
                             const uint32_t n_threads,
                             const uint32_t type,
                             const uint32_t n) {
     
     if(type==1U) {
         uint32_t threadsBlock = 32;
         uint32_t blocksGrid   = (n_threads + threadsBlock - 1) / threadsBlock;
         n_jammers_range_kernel1<<<blocksGrid,threadsBlock>>>(Kth,gamma,h,w,Ln,rho,tf,tr,
                                                              Pt,Frdr,Fp,F,Flen,sig,Ts,Dx,
                                                              Lt,La,Rnj,n_threads);
     }
     else if(type==2U) {
          uint32_t threadsBlock = 256;
          uint32_t blocksGrid   = (n + threadsBlock - 1) / threadsBlock;
          n_jammers_range_kernel2<<<blocksGrid,threadsBlock>>>(Kth,gamma,h,w,Ln,rho,tf,tr,
                                                              Pt,Frdr,Fp,F,Flen,sig,Ts,Dx,
                                                              Lt,La,Rnj,n);
     }
} 



 
                                  
