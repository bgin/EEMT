
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "GMS_cuda_memops.cuh"


//
//	Implementation
//



void copy_int32_cpu_to_gpu(int32_t * __restrict d_ptr, 
			   const int32_t * __restrict h_ptr,
			   const size_t n,
			   int32_t * ierr )
 {
	if(*ierr <= 0) *ierr = 0;
	if( NULL == h_ptr || n <= 0){ //  Host error handling
	   *ierr = -1;
	   return; 
	}
	hipError_t status;

#if (GMS_CUDA_DEBUG_ON) == 1

	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(int32_t)));
	GMS_CUDA_DEBUG_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n * sizeof(int32_t),hipMemcpyHostToDevice));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(int32_t)));
	GMS_CUDA_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(int32_t),hipMemcpyHostToDevice));
#endif
	*ierr = 0;
        return;
Error:
      
	if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	   fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
        else{
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}






void copy_float_cpu_to_gpu(float * __restrict d_ptr,
			   const float * __restrict h_ptr,
			   const size_t n,
			   int32_t * ierr ) {
	if(*ierr <= 0) *ierr = 0;
	if(NULL == h_ptr || n <= 0){
	   *ierr = -1;
	   return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(float)));
	GMS_CUDA_DEBUG_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(float),
					hipMemcpyHostToDevice));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(float)));
	GMS_CUDA_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(float),
					hipMemcpyHostToDevice));
#endif
	*ierr = 0;
        return;
Error:
	if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	   fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
         else {
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}






void copy_double_cpu_to_gpu(double * __restrict d_ptr,
			    const double * __restrict h_ptr,
			    const size_t n,
			    int32_t * ierr ) {
	if(*ierr <= 0) *ierr = 0;
	if(NULL == h_ptr ||
	   0 >= n     ) {
	    *ierr = -1;
	    return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(double)));
	GMS_CUDA_DEBUG_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(double),
					hipMemcpyHostToDevice));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(double)));
	GMS_CUDA_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(double),
					hipMemcpyHostToDevice));
#endif
	*ierr = 0;
        return;
Error:
      if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	 fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
        else {
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}

void copy_complex4_cpu_to_gpu(hipComplex * __restrict d_ptr,
                              hipComplex * __restrict h_ptr,
                              const size_t n,
                              int32_t * ierr) {
       if(*ierr <= 0) *ierr = 0;
       if(NULL == h_ptr ||
	   0 >= n     ) {
	    *ierr = -1;
	    return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(hipComplex)));
	GMS_CUDA_DEBUG_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(hipComplex),
					hipMemcpyHostToDevice));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(hipComplex)));
	GMS_CUDA_CHECK(hipMemcpy((void*)&d_ptr[0],(void*)&h_ptr[0],n*sizeof(hipComplex),
					hipMemcpyHostToDevice));
#endif
	*ierr = 0;
        return;
Error:
        if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
         else {
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}



//
//	Allocate memory on GPU.
//

void alloc_int32_gpu(int32_t * __restrict d_ptr,
                     const size_t n,
		     int32_t * ierr ) {
	if(*ierr < 0) *ierr = 0;
	if(0 >= n) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(int32_t)));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(int32_t)));
#endif
	*ierr = 0;
        return;
Error:
	if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	   fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
        else {
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}







void alloc_float_gpu(float * __restrict d_ptr,
		     const size_t n,
		     int32_t * ierr ) {
	if(*ierr < 0) *ierr = 0;
	if(0 >= n ) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(float)));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(float)));
#endif
	*ierr = 0;
        return;
Error:
	if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	   fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
         else {
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}





void alloc_double_gpu(double * __restrict d_ptr,
		      const size_t n,
		      int32_t * ierr ) {
	if(*ierr < 0) *ierr = 0;
	if(0 >= n ) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(double)));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(double)));
#endif
	*ierr = 0;
        return;
Error:
	
        if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	  fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
          }
          else {
	      if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}

void alloc_complex4_gpu(hipComplex * __restrict d_ptr,
                        const size_t n,
                        int32_t * ierr) {
    if(*ierr < 0) *ierr = 0;
    if(0 >= n ) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(hipComplex)));
#else
	GMS_CUDA_CHECK(hipMalloc((void**)&d_ptr,n*sizeof(hipComplex)));
#endif
	*ierr = 0;
        return;
Error:
       if(__builtin_expect(status==hipErrorOutOfMemory,0)){
	  fatal_gpu_error(__PRETTY_FUNCTION__,
		            status);
        }
        else {
	   if(d_ptr) hipFree(d_ptr);
	      *ierr = -2;
             return;
       } 
}



//
// GPU to CPU memory copy routines
//


//
// Copy array  of int32_t from GPU to CPU.
//

void copy_int32_gpu_to_cpu(const int32_t * __restrict d_ptr,
			   int32_t * __restrict h_ptr,
			   const int32_t n,
			   int32_t * ierr ) {
	if(*ierr < 0) *ierr = 0;
	if(0 >= n) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(int32_t),
					hipMemcpyDeviceToHost));
	
#else
	GMS_CUDA_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(int32_t),
					hipMemcpyDeviceToHost));
	
#endif
        *ierr = 0;
         return;
Error:
	
	*ierr = -2;
	return;
}



void copy_float_gpu_to_cpu(const float * __restrict d_ptr,
			   float * __restrict h_ptr,
			   const int32_t n,
			   int32_t * ierr ) {
	if(*ierr < 0) *ierr = 0;
	if(0 >= n) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(float),
					hipMemcpyDeviceToHost));
	
#else
	GMS_CUDA_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(float),
					hipMemcpyDeviceToHost));
	
#endif
        *ierr = 0;
        return;
Error:
	
	*ierr = -2;
	return;
}



void copy_double_gpu_to_cpu(const double * __restrict d_ptr,
			    double * __restrict h_ptr,
			    const int32_t n,
			    int32_t * ierr ) {
	if(*ierr < 0) *ierr = 0;
	if(0 >= n) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(double),
					hipMemcpyDeviceToHost));
	
#else
	GMS_CUDA_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(double),
			   hipMemcpyDeviceToHost));
	
#endif
        *ierr = 0;
Error:
	
	*ierr = -2;
	 return;
}


void copy_complex4_gpu_to_cpu(hipComplex * __restrict d_ptr,
                              hipComplex * __restrict h_ptr,
                              const size_t n,
                              int32_t ierr) {
        if(*ierr < 0) *ierr = 0;
	if(0 >= n) {

		*ierr = -1;
		return;
	}
	hipError_t status;
#if (GMS_CUDA_DEBUG_ON) == 1
	GMS_CUDA_DEBUG_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(hipComplex),
					hipMemcpyDeviceToHost));
	
#else
	GMS_CUDA_CHECK(hipMemcpy(&h_ptr[0],&d_ptr[0],n*sizeof(hipComplex),
			   hipMemcpyDeviceToHost));
	
#endif
         *ierr = 0;
         return;
Error:
	
	*ierr = -2;
	 return;
} 

























